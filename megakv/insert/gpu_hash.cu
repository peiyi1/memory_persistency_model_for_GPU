#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2015 Kai Zhang (kay21s@gmail.com)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE.
 */

#include <stdint.h>
#include <stdio.h>
#include <assert.h>
#include "gpu_hash.h"

#include "nvm_til.h"

#include "nvmb.cu"
#include "nvmo.cu"
#include "nvmu.cu"

__global__ void kernel_l2wb(void){
        L2WB;
        MEM_FENCE;
}
__global__ void kernel_l2wb_pct(void){
        L2WB;
        MEM_FENCE;
         PCOMMIT; MEM_FENCE;
}

__global__ void hash_insert_cuckoo(
		bucket_t		*hash_table,
		ielem_t			**blk_input,
		int				*blk_elem_num)
{
	ielem_t *in = blk_input[blockIdx.x];
	int total_elem_num = blk_elem_num[blockIdx.x];
	// 16 threads to cooperate for one element
	int step = blockDim.x >> ELEM_NUM_P;
	int idx = threadIdx.x;

	hash_t hash, second_hash;
	loc_t loc, new_loc;
	sign_t sig, new_sig;

	int id;
	int cuckoo_num;
	bucket_t *b;
	int chosen_simd;
	int ballot, ml_mask;

	int simd_lane = idx & ((1 << ELEM_NUM_P) - 1);
	int elem_id = idx >> ELEM_NUM_P;
	int bit_move = idx & (((1 << (5 - ELEM_NUM_P)) - 1) << ELEM_NUM_P);

	for (id = elem_id; id < total_elem_num; id += step) {
		ielem_t *elem = &(in[id]);

		if (elem->sig == 0 && elem->loc == 0) {
			printf("error, all is zero\n");
			continue;
		}

		sig = elem->sig;
		hash = elem->hash;
		loc = elem->loc;

		b = &(hash_table[hash & HASH_MASK]);

		/*=====================================================================
		 * The double __syncthreads() seems useless in else, this is to match the two in
		 * if (chosen_simd == simd_lane). As is stated in the paper <Demystifying GPU 
		 * Microarchitecture through Microbenchmarking>, the __syncthreads() will not go
		 * wrong if not all threads in one wrap reach it. However, the wraps in the same
		 * block need to reach a __syncthreads(), even if they are not on the same line */
		/* Check for same signatures in two bucket */
		ballot = __ballot_sync(FULL_MASK,b->sig[simd_lane] == sig);
		/* first half warp(0~15 threads), bit_move = 0
		 * for second half warp(16~31 threads), bit_move = 16 */
		ballot = (ballot >> bit_move) & ((1 << ELEM_NUM) - 1);
		if (ballot != 0) {
			chosen_simd = (__ffs(ballot) - 1) & ((1 << ELEM_NUM_P) - 1);
			if (simd_lane == chosen_simd) {
				b->loc[simd_lane] = loc;
			}
			continue;
		}

		/*=====================================================================*/
		/* Next we try to insert, the while loop breaks if ballot == 0, and the 
		 * __syncthreads() in the two loops match if the code path divergent between
		 * the warps in a block. Or some will terminate, or process the next element. 
		 * FIXME: if some wrap go to process next element, some stays here, will this
		 * lead to mismatch in __syncthreads()? If it does, we should launch one thread
		 * for each element. God knows what nVidia GPU will behave. FIXME;
		 * Here we write b->loc, and the above code also write b->loc. This will not
		 * lead to conflicts, because here all the signatures are 0, while the aboves
		 * are all non-zero */

		/* Major Location : use last 4 bits of signature */
		ml_mask = (1 << (sig & ((1 << ELEM_NUM_P) - 1))) - 1;
		/* find the empty slot for insertion */
		while (1) {
			ballot = __ballot_sync(FULL_MASK,b->sig[simd_lane] == 0);
			ballot = (ballot >> bit_move) & ((1 << ELEM_NUM) - 1);
			/* 1010|0011 => 0000 0011 1010 0000, 16 bits to 32 bits*/
			ballot = ((ballot & ml_mask) << 16) | ((ballot & ~(ml_mask)));
			if (ballot != 0) {
				chosen_simd = (__ffs(ballot) - 1) & ((1 << ELEM_NUM_P) - 1);
				if (simd_lane == chosen_simd) {
					b->sig[simd_lane] = sig;
				}
			}

			__syncthreads();

			if (ballot != 0) {
				if (b->sig[chosen_simd] == sig) {
					if (simd_lane == chosen_simd) {
						b->loc[simd_lane] = loc;
					}
					goto finish;
				}
			} else {
				break;
			}
		}


		/* ==== try next bucket ==== */
		cuckoo_num = 0;

cuckoo_evict:
		second_hash = (((hash ^ sig) & BLOCK_HASH_MASK) 
				| (hash & ~BLOCK_HASH_MASK)) & HASH_MASK; 
		b = &(hash_table[second_hash]);
		/*=====================================================================*/
		/* Check for same signatures in two bucket */
		ballot = __ballot_sync(FULL_MASK,b->sig[simd_lane] == sig);
		/* first half warp(0~15 threads), bit_move = 0
		 * for second half warp(16~31 threads), bit_move = 16 */
		ballot = (ballot >> bit_move) & ((1 << ELEM_NUM) - 1);
		if (0 != ballot) {
			chosen_simd = (__ffs(ballot) - 1) & ((1 << ELEM_NUM_P) - 1);
			if (simd_lane == chosen_simd) {
				b->loc[simd_lane] = loc;
			}
			continue;
		}

		while (1) {
			ballot = __ballot_sync(FULL_MASK,b->sig[simd_lane] == 0);
			ballot = (ballot >> bit_move) & ((1 << ELEM_NUM) - 1);
			ballot = ((ballot & ml_mask) << 16) | ((ballot & ~(ml_mask)));
			if (ballot != 0) {
				chosen_simd = (__ffs(ballot) - 1) & ((1 << ELEM_NUM_P) - 1);
			} else {
				/* No available slot.
				 * Get a Major location between 0 and 15 for insertion */
				chosen_simd = elem->sig & ((1 << ELEM_NUM_P) - 1);
				if (cuckoo_num < MAX_CUCKOO_NUM) {
					/* record the signature to be evicted */
					new_sig = b->sig[chosen_simd];
					new_loc = b->loc[chosen_simd];
				}
			}
			
			/* synchronize before the signature is written by others */
			__syncthreads();

			if (ballot != 0) {
				if (simd_lane == chosen_simd) {
					b->sig[simd_lane] = sig;
				}
			} else {
				/* two situations to handle: 1) cuckoo_num < MAX_CUCKOO_NUM,
				 * replace one element, and reinsert it into its alternative bucket.
				 * 2) cuckoo_num >= MAX_CUCKOO_NUM.
				 * The cuckoo evict exceed the maximum insert time, replace the element.
				 * In each case, we write the signature first.*/
				if (simd_lane == chosen_simd) {
					b->sig[simd_lane] = sig;
				}
			}

			__syncthreads();

			if (ballot != 0) {
				/* write the empty slot or try again when conflict */
				if (b->sig[chosen_simd] == sig) {
					if (simd_lane == chosen_simd) {
						b->loc[simd_lane] = loc;
					}
					goto finish;
				}
			} else {
				if (cuckoo_num < MAX_CUCKOO_NUM) {
					cuckoo_num ++;
					if (b->sig[chosen_simd] == sig) {
						if (simd_lane == chosen_simd) {
							b->loc[simd_lane] = loc;
						}
						sig = new_sig;
						loc = new_loc;
						goto cuckoo_evict;
					} else {
						/* if there is conflict when writing the signature,
						 * it has been replaced by another one. Reinserting
						 * the element is meaningless, because it will evict
						 * the one that is just inserted. Only one will survive,
						 * we just give up the failed one */
						goto finish;
					}
				} else {
					/* exceed the maximum insert time, evict one */
					if (b->sig[chosen_simd] == sig) {
						if (simd_lane == chosen_simd) {
							b->loc[simd_lane] = loc;
						}
					}
					/* whether or not succesfully inserted, finish */
					goto finish;
				}
			}
		}

finish:
		;
		//now we get to the next element
	}
	return;
}

/* num_blks is the array size of blk_input and blk_output */
extern "C" void gpu_hash_insert(
		bucket_t	*hash_table,
		ielem_t		**blk_input,
		int			*blk_elem_num,
		int			num_blks,
		hipStream_t stream)
{
	int threads_per_blk = 1024;
	//printf("hash_insert: num_blks %d, threads_per_blk %d\n", num_blks, threads_per_blk);

	// prefer L1 cache rather than shared cache
	//void (*funcPtr)(bucket_t *, ielem_t **, loc_t **, int *);
	//funcPtr = hash_insert;
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(*funcPtr), hipFuncCachePreferL1);
	assert(ELEM_NUM_P < 5 && ELEM_NUM_P > 0);

syncLapTimer st;
st.lap_start();
	if (stream == 0) {
		if (nvm_opt == 'a')
		hash_insert_cuckoo<<<num_blks, threads_per_blk>>>(
			hash_table, blk_input, blk_elem_num);
		else if (nvm_opt == 'b')
		hash_insert_cuckoo_nvmb<<<num_blks, threads_per_blk>>>(
			hash_table, blk_input, blk_elem_num);
		else if (nvm_opt == 'o')
		hash_insert_cuckoo_nvmo<<<num_blks, threads_per_blk>>>(
			hash_table, blk_input, blk_elem_num);
		else if (nvm_opt == 'u')
		hash_insert_cuckoo_nvmu<<<num_blks, threads_per_blk>>>(
			hash_table, blk_input, blk_elem_num);
		
		else if (nvm_opt == 'k'){
		hash_insert_cuckoo<<<num_blks, threads_per_blk>>>(
                        hash_table, blk_input, blk_elem_num);
		hipDeviceSynchronize();
                kernel_l2wb <<< num_blks, threads_per_blk>>>();
		}
		else if (nvm_opt == 'l'){
		hash_insert_cuckoo<<<num_blks, threads_per_blk>>>(
                        hash_table, blk_input, blk_elem_num);
                hipDeviceSynchronize();
		kernel_l2wb_pct <<< num_blks, threads_per_blk>>>();
		}
	}else{
                if (nvm_opt == 'a')
                hash_insert_cuckoo<<<num_blks, threads_per_blk, 0, stream>>>(
                        hash_table, blk_input, blk_elem_num);
                else if (nvm_opt == 'b')
                hash_insert_cuckoo_nvmb<<<num_blks, threads_per_blk, 0, stream>>>(
                        hash_table, blk_input, blk_elem_num);
                else if (nvm_opt == 'o')
                hash_insert_cuckoo_nvmo<<<num_blks, threads_per_blk, 0, stream>>>(
                        hash_table, blk_input, blk_elem_num);
                else if (nvm_opt == 'u')
                hash_insert_cuckoo_nvmu<<<num_blks, threads_per_blk, 0, stream>>>(
                        hash_table, blk_input, blk_elem_num);

		else if (nvm_opt == 'k'){
		hash_insert_cuckoo<<<num_blks, threads_per_blk, 0, stream>>>(
                        hash_table, blk_input, blk_elem_num);
                hipDeviceSynchronize();
                kernel_l2wb <<< num_blks, threads_per_blk, 0, stream>>>();
                }
                else if (nvm_opt == 'l'){
                hash_insert_cuckoo<<<num_blks, threads_per_blk, 0, stream>>>(
                        hash_table, blk_input, blk_elem_num);
                hipDeviceSynchronize();
                kernel_l2wb_pct <<< num_blks, threads_per_blk, 0, stream>>>();
                }

	}
st.lap_end();
st.print_total_us("total_us");
st.print_avg_usec("gen_hists", num_blks);
	return;
}

