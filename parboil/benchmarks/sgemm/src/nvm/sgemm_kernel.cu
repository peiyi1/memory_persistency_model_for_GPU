#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/* 
 * Kernel of dense matrix-matrix multiplication kernel.
 * The algorithm is based on CUDA sgemm code from Vasily Volkov
 * at UC Berkeley.
 */

#include "nvm_util.h"
#define CHECK_ERROR(errorMessage) {                                    \
  hipError_t err = hipGetLastError();                                    \
  if( hipSuccess != err) {                                                \
    fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",    \
	errorMessage, __FILE__, __LINE__, hipGetErrorString( err) );\
    exit(EXIT_FAILURE);                                                  \
  }                                                                        \
}

// CML x RML = CML, baseline version, 510FLOP/s on Fermi
/* Pseudo code
for i < M ; i += 64   // thread block.x
 for j < N; j += 16   // thread block.y
  for tx = 0; tx < 16; tx++ // thread index x; tile of M loop
  for ty = 0; ty < 4 ; ty++ // thread index y; tile of M loop

  for m < 16; m += 1;
     c[m] = 0.0f

  for k < K; k += 4   // seq

   b[ty][tx] = B[k+ty][j+tx]

   for l < 4; l +=1   // seq
    for m < 16; m +=1 // seq
      c[m] += A[i+ty*16+tx][k+l]+b[l][m]

*/

// Parameters of tile sizes
#define TILE_N 16 
#define TILE_TB_HEIGHT 8
#define TILE_M (TILE_N*TILE_TB_HEIGHT)

#define STC_WT(i) ST_WT_FLOAT(&C[t+(i)*ldc], C[t+(i)*ldc] * beta + alpha * c[(i)]); 

#include "nvm_kernels.cu"
#include "nvm_log_kernels.cu"

__global__ void mysgemmNT( const float *A, int lda, const float *B, int ldb, float* C, int ldc, int k, float alpha, float beta )
{
    // Partial results 
    float c[TILE_N];
    for (int i=0; i < TILE_N; i++)
	c[i] = 0.0f;
    int mid = threadIdx.y * blockDim.x + threadIdx.x; //flattened id
    int m = blockIdx.x * TILE_M + mid;
    int n = blockIdx.y * TILE_N + threadIdx.x;
    __shared__ float b_s[TILE_TB_HEIGHT][TILE_N];
    for (int i = 0; i < k; i+=TILE_TB_HEIGHT) {
	float a; 
	b_s[threadIdx.y][threadIdx.x]=B[n + (i+threadIdx.y)*ldb];
	__syncthreads();
	for (int j = 0; j < TILE_TB_HEIGHT; j++) {
	    a = A[m + (i+j)*lda];
	    for (int kk = 0; kk < TILE_N; kk++)
		c[kk] += a * b_s[j][kk];

	}
	__syncthreads();
    }
    int t = ldc*blockIdx.y * TILE_N + m;
    for (int i = 0; i < TILE_N; i++) {
	C[t+i*ldc] = C[t+i*ldc] * beta + alpha * c[i];
    }
}


__global__ void mysgemmNT_nvmb( const float *A, int lda, const float *B, int ldb, float* C, int ldc, int k, float alpha, float beta )
{
    // Partial results 
    float c[TILE_N];
    for (int i=0; i < TILE_N; i++)
	c[i] = 0.0f;
    int mid = threadIdx.y * blockDim.x + threadIdx.x; //flattened id
    int m = blockIdx.x * TILE_M + mid;
    int n = blockIdx.y * TILE_N + threadIdx.x;
    __shared__ float b_s[TILE_TB_HEIGHT][TILE_N];
    for (int i = 0; i < k; i+=TILE_TB_HEIGHT) {
	float a; 
	b_s[threadIdx.y][threadIdx.x]=B[n + (i+threadIdx.y)*ldb];
	__syncthreads();
	for (int j = 0; j < TILE_TB_HEIGHT; j++) {
	    a = A[m + (i+j)*lda];
	    for (int kk = 0; kk < TILE_N; kk++)
		c[kk] += a * b_s[j][kk];

	}
	__syncthreads();
    }
    int t = ldc*blockIdx.y * TILE_N + m;
    /*
    for (int i = 0; i < TILE_N; i++) {
      //C[t+i*ldc] = C[t+i*ldc] * beta + alpha * c[i];
      float result = C[t+i*ldc] * beta + alpha * c[i];
      //C[t+i*ldc] = result;
      ST_WT_FLOAT(&C[t+i*ldc], result);
      MEM_FENCE;
    }
    */
    // have to manually unroll it to avoid register spilling
    STC_WT(0); SFENCE;
    STC_WT(1); SFENCE;
    STC_WT(2); SFENCE;
    STC_WT(3); SFENCE;
    STC_WT(4); SFENCE;
    STC_WT(5); SFENCE;
    STC_WT(6); SFENCE;
    STC_WT(7); SFENCE;
    STC_WT(8); SFENCE;
    STC_WT(9); SFENCE;
    STC_WT(10); SFENCE;
    STC_WT(11); SFENCE;
    STC_WT(12); SFENCE;
    STC_WT(13); SFENCE;
    STC_WT(14); SFENCE;
    STC_WT(15); SFENCE;
}


__global__ void mysgemmNT_nvmd( const float *A, int lda, const float *B, int ldb, float* C, int ldc, int k, float alpha, float beta )
{
    // Partial results 
    float c[TILE_N];
    for (int i=0; i < TILE_N; i++)
	c[i] = 0.0f;
    int mid = threadIdx.y * blockDim.x + threadIdx.x; //flattened id
    int m = blockIdx.x * TILE_M + mid;
    int n = blockIdx.y * TILE_N + threadIdx.x;
    __shared__ float b_s[TILE_TB_HEIGHT][TILE_N];
    for (int i = 0; i < k; i+=TILE_TB_HEIGHT) {
	float a; 
	b_s[threadIdx.y][threadIdx.x]=B[n + (i+threadIdx.y)*ldb];
	__syncthreads();
	for (int j = 0; j < TILE_TB_HEIGHT; j++) {
	    a = A[m + (i+j)*lda];
	    for (int kk = 0; kk < TILE_N; kk++)
		c[kk] += a * b_s[j][kk];

	}
	__syncthreads();
    }
    int t = ldc*blockIdx.y * TILE_N + m;
    STC_WT(0); 
    STC_WT(1); 
    STC_WT(2); 
    STC_WT(3); 
    STC_WT(4); 
    STC_WT(5); 
    STC_WT(6); 
    STC_WT(7); 
    STC_WT(8); 
    STC_WT(9); 
    STC_WT(10); 
    STC_WT(11); 
    STC_WT(12); 
    STC_WT(13); 
    STC_WT(14); 
    STC_WT(15); 
    SFENCE;
}

void regtileSgemm( char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc )
{
  if ((transa != 'N') && (transa != 'n')) {
    std::cerr << "unsupported value of 'transa' in regtileSgemm()" << std::endl;
    return;
  }
  
  if ((transb != 'T') && (transb != 't')) {
    std::cerr << "unsupported value of 'transb' in regtileSgemm()" << std::endl;
    return;
  }
  
  // In this code we assume the matrix sizes are multiple of tile size
  if ((m%TILE_M) || (n%TILE_N)) {
    std::cerr << "unsupported size of matrix. m should be multiple of " << TILE_M
      << "; n should be multiple of " << TILE_N << std::endl;
  }
  syncLapTimer st;
  
  dim3 grid( m/TILE_M, n/TILE_N ), threads( TILE_N, TILE_TB_HEIGHT );
  st.lap_start();
  if (nvm_opt == 'a')
    mysgemmNT<<<grid, threads>>>( A, lda, B, ldb, C, ldc, k, alpha, beta);
  else if (nvm_opt == 'b')
    mysgemmNT_nvmb<<<grid, threads>>>( A, lda, B, ldb, C, ldc, k, alpha, beta);
  else if (nvm_opt == 'd')
    mysgemmNT_nvmd<<<grid, threads>>>( A, lda, B, ldb, C, ldc, k, alpha, beta);
  else if (nvm_opt == 'g')
    mysgemmNT_nvmg<<<grid, threads>>>( A, lda, B, ldb, C, ldc, k, alpha, beta);
  else if (nvm_opt == 'o')
    mysgemmNT_nvmo<<<grid, threads>>>( A, lda, B, ldb, C, ldc, k, alpha, beta);
  else if (nvm_opt == 'u')
    mysgemmNT_nvmu<<<grid, threads>>>( A, lda, B, ldb, C, ldc, k, alpha, beta);
  else if (nvm_opt == 'q')
    mysgemmNT_nvmq<<<grid, threads>>>( A, lda, B, ldb, C, ldc, k, alpha, beta);
  else if (nvm_opt == 'w')
    mysgemmNT_nvmw<<<grid, threads>>>( A, lda, B, ldb, C, ldc, k, alpha, beta);
  else if (nvm_opt == '1')
    mysgemmNT_nvm1<<<grid, threads>>>( A, lda, B, ldb, C, ldc, k, alpha, beta);
  else if (nvm_opt == '2')
    mysgemmNT_nvm2<<<grid, threads>>>( A, lda, B, ldb, C, ldc, k, alpha, beta);
  else if (nvm_opt == '3')
    mysgemmNT_nvm3<<<grid, threads>>>( A, lda, B, ldb, C, ldc, k, alpha, beta);
  else if (nvm_opt == '4')
    mysgemmNT_nvm4<<<grid, threads>>>( A, lda, B, ldb, C, ldc, k, alpha, beta);
  /*
  else if (nvm_opt == 'j')
    mysgemmNT_nvmj<<<grid, threads>>>( A, lda, B, ldb, C, ldc, k, alpha, beta);
  else if (nvm_opt == 'l')
    mysgemmNT_nvml<<<grid, threads>>>( A, lda, B, ldb, C, ldc, k, alpha, beta);
  */
  st.lap_end();
  st.print_avg_usec("mysegemmNT", grid);
  CHECK_ERROR("mySgemm");

}

