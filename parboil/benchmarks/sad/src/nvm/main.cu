#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <inttypes.h>
#include <parboil.h>
#include <hip/hip_runtime.h>

#include "sad.h"
#include "sad4.h"
#include "largerBlocks.h"
#include "file.h"
#include "image.h"

#define CUDA_ERRCK \
  {hipError_t err = hipGetLastError(); \
    if (err) fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err)); \
  }

static unsigned short *
load_sads(char *filename);
static void
write_sads(char *filename,
	   int image_width_macroblocks,
	   int image_height_macroblocks,
	   unsigned short *sads);
static void
write_sads_directly(char *filename,
		    int width,
		    int height,
		    unsigned short *sads);

__global__ void kernel_l2wb(void){
	L2WB;
	MEM_FENCE;
}
__global__ void kernel_l2wb_pct(void){
        L2WB;
        MEM_FENCE;
	 PCOMMIT; MEM_FENCE;
}
/* FILE I/O */

unsigned short *
load_sads(char *filename)
{
  FILE *infile;
  unsigned short *sads;
  int w;
  int h;
  int sads_per_block;

  infile = fopen(filename, "r");

  if (!infile)
    {
      fprintf(stderr, "Cannot find file '%s'\n", filename);
      exit(-1);
    }

  /* Read image dimensions (measured in macroblocks) */
  w = read16u(infile);
  h = read16u(infile);

  /* Read SAD values.  Only interested in the 4x4 SAD values, which are
   * at the end of the file. */
  sads_per_block = MAX_POS_PADDED * (w * h);
  fseek(infile, 25 * sads_per_block * sizeof(unsigned short), SEEK_CUR);

  sads = (unsigned short *)malloc(sads_per_block * 16 * sizeof(unsigned short));
  fread(sads, sizeof(unsigned short), sads_per_block * 16, infile);
  fclose(infile);

  return sads;
}

/* Compare the reference SADs to the expected SADs.
 */
void
check_sads(unsigned short *sads_reference,
	   unsigned short *sads_computed,
	   int image_size_macroblocks)
{
  int block;

  /* Check the 4x4 SAD values.  These are in sads_reference.
   * Ignore the data at the beginning of sads_computed. */
  sads_computed += 25 * MAX_POS_PADDED * image_size_macroblocks;

  for (block = 0; block < image_size_macroblocks; block++)
    {
      int subblock;

      for (subblock = 0; subblock < 16; subblock++)
	{
	  int sad_index;

	  for (sad_index = 0; sad_index < MAX_POS; sad_index++)
	    {
	      int index =
		(block * 16 + subblock) * MAX_POS_PADDED + sad_index;

	      if (sads_reference[index] != sads_computed[index])
		{
#if 0
		  /* Print exactly where the mismatch was seen */
		  printf("M %3d %2d %4d (%d = %d)\n", block, subblock, sad_index, sads_reference[index], sads_computed[index]);
#else
		  goto mismatch;
#endif
		}
	    }
	}
    }

  printf("Success.\n");
  return;

 mismatch:
  printf("Computed SADs do not match expected values.\n");
}

/* Extract the SAD data for a particular block type for a particular
 * macroblock from the array of SADs of that block type. */
static inline void
write_subblocks(FILE *outfile, unsigned short *subblock_array, int macroblock,
		int count)
{
  int block;
  int pos;

  for (block = 0; block < count; block++)
    {
      unsigned short *vec = subblock_array +
	(block + macroblock * count) * MAX_POS_PADDED;

      /* Write all SADs for this sub-block */
      for (pos = 0; pos < MAX_POS; pos++)
	write16u(outfile, *vec++);
    }
}

/* Write some SAD data to a file for output checking.
 *
 * All SAD values for six rows of macroblocks are written.
 * The six rows consist of the top two, middle two, and bottom two image rows.
 */
void
write_sads(char *filename,
	   int mb_width,
	   int mb_height,
	   unsigned short *sads)
{
  FILE *outfile = fopen(filename, "w");
  int mbs = mb_width * mb_height;
  int row_indir;
  int row_indices[6] = {0, 1,
			mb_height / 2 - 1, mb_height / 2,
			mb_height - 2, mb_height - 1};

  if (outfile == NULL)
    {
      fprintf(stderr, "Cannot open output file\n");
      exit(-1);
    }

  /* Write the number of output macroblocks */
  write32u(outfile, mb_width * 6);

  /* Write zeros */
  write32u(outfile, 0);

  /* Each row */
  for (row_indir = 0; row_indir < 6; row_indir++)
    {
      int row = row_indices[row_indir];

      /* Each block in row */
      int block;
      for (block = mb_width * row; block < mb_width * (row + 1); block++)
	{
	  int blocktype;

	  /* Write SADs for all sub-block types */
	  for (blocktype = 1; blocktype <= 7; blocktype++)
	    write_subblocks(outfile,
			    sads + SAD_TYPE_IX(blocktype, mbs),
			    block,
			    SAD_TYPE_CT(blocktype));
	}
    }

  fclose(outfile);
}

/* FILE I/O for debugging */

static void
write_sads_directly(char *filename,
		    int width,
		    int height,
		    unsigned short *sads)
{
  FILE *f = fopen(filename, "w");
  int n;

  write16u(f, width);
  write16u(f, height);
  for (n = 0; n < 41 * MAX_POS_PADDED * (width * height); n++) {
    write16u(f, sads[n]);
  }
  fclose(f);
}

static void
print_test_sad_vector(unsigned short *base, int macroblock, int count)
{
  int n;
  int searchpos = 17*33+17;
  for (n = 0; n < count; n++)
    printf(" %d", base[(count * macroblock + n) * MAX_POS_PADDED + searchpos]);
}

static void
print_test_sads(unsigned short *sads_computed,
		int mbs)
{
  int macroblock = 5;
  int blocktype;

  for (blocktype = 1; blocktype <= 7; blocktype++)
    {
      printf("%d:", blocktype);
      print_test_sad_vector(sads_computed + SAD_TYPE_IX(blocktype, mbs),
			    macroblock, SAD_TYPE_CT(blocktype));
      puts("\n");
    }
}

/* MAIN */
char nvm_opt;

int
main(int argc, char **argv)
{
  nvm_opt = *argv[--argc];
  
  struct image_i16 *ref_image;
  struct image_i16 *cur_image;
  unsigned short *sads_computed; /* SADs generated by the program */

  int image_size_bytes;
  int image_width_macroblocks, image_height_macroblocks;
  int image_size_macroblocks;

  struct pb_TimerSet timers;
  struct pb_Parameters *params;

  pb_InitializeTimerSet(&timers);
  params = pb_ReadParameters(&argc, argv);

  if (pb_Parameters_CountInputs(params) != 2)
    {
      fprintf(stderr, "Expecting two input filenames\n");
      exit(-1);
    }

  /* Read input files */
  pb_SwitchToTimer(&timers, pb_TimerID_IO);
  ref_image = load_image(params->inpFiles[0]);
  cur_image = load_image(params->inpFiles[1]);
  pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

  if ((ref_image->width != cur_image->width) ||
      (ref_image->height != cur_image->height))
    {
      fprintf(stderr, "Input images must be the same size\n");
      exit(-1);
    }
  if ((ref_image->width % 16) || (ref_image->height % 16))
    {
      fprintf(stderr, "Input image size must be an integral multiple of 16\n");
      exit(-1);
    }

  /* Compute parameters, allocate memory */
  image_size_bytes = ref_image->width * ref_image->height * sizeof(short);
  image_width_macroblocks = ref_image->width >> 4;
  image_height_macroblocks = ref_image->height >> 4;
  image_size_macroblocks = image_width_macroblocks * image_height_macroblocks;
  
  sads_computed = (unsigned short *)
    malloc(41 * MAX_POS_PADDED * image_size_macroblocks * sizeof(short));

  /* Run the kernel code */
  {
    struct hipArray *ref_ary;  /* Reference image on the device */
    short *d_cur_image;         /* Current image on the device */
    unsigned short *d_sads;     /* SADs on the device */
    dim3 macroblock_grid(image_width_macroblocks, image_height_macroblocks);

    pb_SwitchToTimer(&timers, pb_TimerID_COPY);
    hipMalloc((void **)&d_cur_image, image_size_bytes);
    CUDA_ERRCK
    hipMallocArray(&ref_ary, &get_ref().channelDesc,
                    ref_image->width, ref_image->height);
    CUDA_ERRCK

    /* Transfer current image to device */
    hipMemcpy(d_cur_image, cur_image->data, image_size_bytes,
               hipMemcpyHostToDevice);
    CUDA_ERRCK

    /* Transfer reference image to device */
    hipMemcpy2DToArray(ref_ary,
                        0, 0,
                        ref_image->data,
                        ref_image->width * sizeof(unsigned short),
                        ref_image->width * sizeof(unsigned short),
                        ref_image->height,
                        hipMemcpyHostToDevice);
    CUDA_ERRCK
    hipBindTextureToArray(get_ref(), ref_ary);
    CUDA_ERRCK

    /* Allocate SAD data on the device */
    hipMalloc((void **)&d_sads, 41 * MAX_POS_PADDED * image_size_macroblocks *
	       sizeof(unsigned short));
    CUDA_ERRCK;
    unsigned short *NVM_klog;
    hipMalloc((void **)&NVM_klog, 41 * MAX_POS_PADDED * image_size_macroblocks *
	       sizeof(unsigned short));
    CUDA_ERRCK
    hipMemset(d_sads, 0, 41 * MAX_POS_PADDED * image_size_macroblocks *
	       sizeof(unsigned short));
    CUDA_ERRCK

    pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);

    printf("width * height = %d\n", image_width_macroblocks * image_height_macroblocks);
    syncLapTimer st_mb, st_8, st_16;
    for (int i = 0; i < 1; i++) {
    // Run the 4x4 kernel
      st_mb.lap_start();
    if (nvm_opt == 'a') {
      mb_sad_calc<<<dim3(CEIL(ref_image->width / 4, THREADS_W),
			 CEIL(ref_image->height / 4, THREADS_H)),
	dim3(CEIL(MAX_POS, POS_PER_THREAD) * THREADS_W * THREADS_H),
	SAD_LOC_SIZE_BYTES>>>
	(d_sads,
	 (unsigned short *)d_cur_image,
	 image_width_macroblocks,
	 image_height_macroblocks);
    } else if (nvm_opt == 'b') {
      mb_sad_calc_nvmb<<<dim3(CEIL(ref_image->width / 4, THREADS_W),
			 CEIL(ref_image->height / 4, THREADS_H)),
	dim3(CEIL(MAX_POS, POS_PER_THREAD) * THREADS_W * THREADS_H),
	SAD_LOC_SIZE_BYTES>>>
	(d_sads,
	 (unsigned short *)d_cur_image,
	 image_width_macroblocks,
	 image_height_macroblocks);
    } else if (nvm_opt == 'd') {
      mb_sad_calc_nvmd<<<dim3(CEIL(ref_image->width / 4, THREADS_W),
			 CEIL(ref_image->height / 4, THREADS_H)),
	dim3(CEIL(MAX_POS, POS_PER_THREAD) * THREADS_W * THREADS_H),
	SAD_LOC_SIZE_BYTES>>>
	(d_sads,
	 (unsigned short *)d_cur_image,
	 image_width_macroblocks,
	 image_height_macroblocks);
    } else if (nvm_opt == 'g') {
      mb_sad_calc_nvmg<<<dim3(CEIL(ref_image->width / 4, THREADS_W),
			 CEIL(ref_image->height / 4, THREADS_H)),
	dim3(CEIL(MAX_POS, POS_PER_THREAD) * THREADS_W * THREADS_H),
	SAD_LOC_SIZE_BYTES>>>
	(d_sads,
	 (unsigned short *)d_cur_image,
	 image_width_macroblocks,
	 image_height_macroblocks);
    } else if (nvm_opt == 'i') {
      mb_sad_calc_nvmi<<<dim3(CEIL(ref_image->width / 4, THREADS_W),
			 CEIL(ref_image->height / 4, THREADS_H)),
	dim3(CEIL(MAX_POS, POS_PER_THREAD) * THREADS_W * THREADS_H),
	SAD_LOC_SIZE_BYTES>>>
	(d_sads,
	 (unsigned short *)d_cur_image,
	 image_width_macroblocks,
	 image_height_macroblocks);
    } else if (nvm_opt == 'o') {
      mb_sad_calc_nvmo<<<dim3(CEIL(ref_image->width / 4, THREADS_W),
			 CEIL(ref_image->height / 4, THREADS_H)),
	dim3(CEIL(MAX_POS, POS_PER_THREAD) * THREADS_W * THREADS_H),
	SAD_LOC_SIZE_BYTES>>>
	(d_sads,
	 (unsigned short *)d_cur_image,
	 image_width_macroblocks,
	 image_height_macroblocks);
    } else if (nvm_opt == 'q') {
      mb_sad_calc_nvmq<<<dim3(CEIL(ref_image->width / 4, THREADS_W),
			 CEIL(ref_image->height / 4, THREADS_H)),
	dim3(CEIL(MAX_POS, POS_PER_THREAD) * THREADS_W * THREADS_H),
	SAD_LOC_SIZE_BYTES>>>
	(d_sads,
	 (unsigned short *)d_cur_image,
	 image_width_macroblocks,
	 image_height_macroblocks);
    } else if (nvm_opt == 'j') {
      mb_sad_calc_nvmj<<<dim3(CEIL(ref_image->width / 4, THREADS_W),
			 CEIL(ref_image->height / 4, THREADS_H)),
	dim3(CEIL(MAX_POS, POS_PER_THREAD) * THREADS_W * THREADS_H),
	SAD_LOC_SIZE_BYTES>>>
	(d_sads,
	 (unsigned short *)d_cur_image,
	 image_width_macroblocks,
	 image_height_macroblocks);
    } else if (nvm_opt == 'l') {
      mb_sad_calc_nvml<<<dim3(CEIL(ref_image->width / 4, THREADS_W),
			 CEIL(ref_image->height / 4, THREADS_H)),
	dim3(CEIL(MAX_POS, POS_PER_THREAD) * THREADS_W * THREADS_H),
	SAD_LOC_SIZE_BYTES>>>
	(d_sads,
	 (unsigned short *)d_cur_image,
	 image_width_macroblocks,
	 image_height_macroblocks);
    } else if (nvm_opt == 'u') {
      mb_sad_calc_nvmu<<<dim3(CEIL(ref_image->width / 4, THREADS_W),
			 CEIL(ref_image->height / 4, THREADS_H)),
	dim3(CEIL(MAX_POS, POS_PER_THREAD) * THREADS_W * THREADS_H),
	SAD_LOC_SIZE_BYTES>>>
	(d_sads,
	 (unsigned short *)d_cur_image,
	 image_width_macroblocks,
	 image_height_macroblocks);
    } else if (nvm_opt == 'w') {
      mb_sad_calc_nvmw<<<dim3(CEIL(ref_image->width / 4, THREADS_W),
			 CEIL(ref_image->height / 4, THREADS_H)),
	dim3(CEIL(MAX_POS, POS_PER_THREAD) * THREADS_W * THREADS_H),
	SAD_LOC_SIZE_BYTES>>>
	(d_sads,
	 (unsigned short *)d_cur_image,
	 image_width_macroblocks,
	 image_height_macroblocks);
    } else if (nvm_opt == '1') {
      mb_sad_calc_nvm1<<<dim3(CEIL(ref_image->width / 4, THREADS_W),
			 CEIL(ref_image->height / 4, THREADS_H)),
	dim3(CEIL(MAX_POS, POS_PER_THREAD) * THREADS_W * THREADS_H),
	SAD_LOC_SIZE_BYTES>>>
	(d_sads,
	 (unsigned short *)d_cur_image,
	 image_width_macroblocks,
	 image_height_macroblocks);
    } else if (nvm_opt == '2') {
      mb_sad_calc_nvm2<<<dim3(CEIL(ref_image->width / 4, THREADS_W),
			 CEIL(ref_image->height / 4, THREADS_H)),
	dim3(CEIL(MAX_POS, POS_PER_THREAD) * THREADS_W * THREADS_H),
	SAD_LOC_SIZE_BYTES>>>
	(d_sads,
	 (unsigned short *)d_cur_image,
	 image_width_macroblocks,
	 image_height_macroblocks);
    } else if (nvm_opt == '3') {
      mb_sad_calc_nvm3<<<dim3(CEIL(ref_image->width / 4, THREADS_W),
			 CEIL(ref_image->height / 4, THREADS_H)),
	dim3(CEIL(MAX_POS, POS_PER_THREAD) * THREADS_W * THREADS_H),
	SAD_LOC_SIZE_BYTES>>>
	(d_sads,
	 (unsigned short *)d_cur_image,
	 image_width_macroblocks,
	 image_height_macroblocks);
    } else if (nvm_opt == '4') {
      mb_sad_calc_nvm4<<<dim3(CEIL(ref_image->width / 4, THREADS_W),
			 CEIL(ref_image->height / 4, THREADS_H)),
	dim3(CEIL(MAX_POS, POS_PER_THREAD) * THREADS_W * THREADS_H),
	SAD_LOC_SIZE_BYTES>>>
	(d_sads,
	 (unsigned short *)d_cur_image,
	 image_width_macroblocks,
	 image_height_macroblocks);
    } else if (nvm_opt == '5') {
      mb_sad_calc_nvm5<<<dim3(CEIL(ref_image->width / 4, THREADS_W),
			 CEIL(ref_image->height / 4, THREADS_H)),
	dim3(CEIL(MAX_POS, POS_PER_THREAD) * THREADS_W * THREADS_H),
	SAD_LOC_SIZE_BYTES>>>
	(d_sads,
	 (unsigned short *)d_cur_image,
	 image_width_macroblocks,
	 image_height_macroblocks);
    } else if (nvm_opt == '6') {
      mb_sad_calc_nvm6<<<dim3(CEIL(ref_image->width / 4, THREADS_W),
			 CEIL(ref_image->height / 4, THREADS_H)),
	dim3(CEIL(MAX_POS, POS_PER_THREAD) * THREADS_W * THREADS_H),
	SAD_LOC_SIZE_BYTES>>>
	(d_sads,
	 (unsigned short *)d_cur_image,
	 image_width_macroblocks,
	 image_height_macroblocks);
    }else{
      mb_sad_calc<<<dim3(CEIL(ref_image->width / 4, THREADS_W),
                         CEIL(ref_image->height / 4, THREADS_H)),
        dim3(CEIL(MAX_POS, POS_PER_THREAD) * THREADS_W * THREADS_H),
        SAD_LOC_SIZE_BYTES>>>
        (d_sads,
         (unsigned short *)d_cur_image,
         image_width_macroblocks,
         image_height_macroblocks);
    }

    st_mb.lap_end();
    CUDA_ERRCK;

    // Run the larger-blocks kernels
    st_8.lap_start();
    if (nvm_opt == 'a') {
      larger_sad_calc_8<<<macroblock_grid, dim3(32, 4)>>>
	(d_sads,
	 image_width_macroblocks,
	 image_height_macroblocks);
    } else if (nvm_opt == 'b') {
      larger_sad_calc_8_nvmb<<<macroblock_grid, dim3(32, 4)>>>
	(d_sads,
	 image_width_macroblocks,
	 image_height_macroblocks);
    } else if (nvm_opt == 'o') {
      larger_sad_calc_8_nvmo<<<macroblock_grid, dim3(32, 4)>>>
	(d_sads,
	 image_width_macroblocks,
	 image_height_macroblocks);
    } else if (nvm_opt == 'u') {
      larger_sad_calc_8_nvmu<<<macroblock_grid, dim3(32, 4)>>>
	(d_sads,
	 image_width_macroblocks,
	 image_height_macroblocks);
    } else if (nvm_opt == 'f') {
      hipMemcpy(NVM_klog, d_sads, 41 * MAX_POS_PADDED * image_size_macroblocks * sizeof(unsigned short), hipMemcpyDeviceToDevice);
      hipDeviceSynchronize();
      larger_sad_calc_8<<<macroblock_grid, dim3(32, 4)>>>
	(d_sads,
	 image_width_macroblocks,
	 image_height_macroblocks);
    } else if (nvm_opt == 'k'){
	larger_sad_calc_8<<<macroblock_grid, dim3(32, 4)>>>
        (d_sads,
         image_width_macroblocks,
         image_height_macroblocks);
	hipDeviceSynchronize();
	kernel_l2wb<<<macroblock_grid, dim3(32, 4)>>>();
    }else if (nvm_opt == 'l'){
        larger_sad_calc_8<<<macroblock_grid, dim3(32, 4)>>>
        (d_sads,
         image_width_macroblocks,
         image_height_macroblocks);
        hipDeviceSynchronize();
        kernel_l2wb_pct<<<macroblock_grid, dim3(32, 4)>>>();
    }else if (nvm_opt == 'm') {
      hipMemcpy(NVM_klog, d_sads, 41 * MAX_POS_PADDED * image_size_macroblocks * sizeof(unsigned short), hipMemcpyDeviceToDevice);
      hipDeviceSynchronize();
      larger_sad_calc_8<<<macroblock_grid, dim3(32, 4)>>>
        (d_sads,
         image_width_macroblocks,
         image_height_macroblocks);
        hipDeviceSynchronize();
        kernel_l2wb<<<macroblock_grid, dim3(32, 4)>>>();

    } 
    st_8.lap_end();
    CUDA_ERRCK;
    exit(0);
    st_16.lap_start();
/*    if (nvm_opt == 'a') {
      larger_sad_calc_16<<<macroblock_grid, dim3(32, 1)>>>
	(d_sads,
	 image_width_macroblocks,
	 image_height_macroblocks);
    }
*/
    st_16.lap_end();
    CUDA_ERRCK;
    } // for 1000

    st_mb.print_avg_usec("mb_sad_calc", CEIL(ref_image->width / 4, THREADS_W) * CEIL(ref_image->height / 4, THREADS_H));
    st_8.print_avg_usec("larger_sad_calc_8", DIM3_NTBS(macroblock_grid));
    st_16.print_avg_usec("larger_sad_calc_16", DIM3_NTBS(macroblock_grid));
    pb_SwitchToTimer(&timers, pb_TimerID_COPY);

    /* Transfer SAD data to the host */
    hipMemcpy(sads_computed,// + 25 * MAX_POS_PADDED * image_size_macroblocks,
	       d_sads,// + 25 * MAX_POS_PADDED * image_size_macroblocks,
	       41 * MAX_POS_PADDED * image_size_macroblocks * sizeof(unsigned short)
,
           hipMemcpyDeviceToHost);
    CUDA_ERRCK

    /* Free GPU memory */
    hipFree(d_sads);
    CUDA_ERRCK
    hipUnbindTexture(get_ref());
    CUDA_ERRCK
    hipFreeArray(ref_ary);
    CUDA_ERRCK
    hipFree(d_cur_image);
    CUDA_ERRCK

    pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
  }

  /* Print output */
  if (params->outFile)
    {
      pb_SwitchToTimer(&timers, pb_TimerID_IO);
      write_sads(params->outFile,
		 image_width_macroblocks,
		 image_height_macroblocks,
		 sads_computed);
      pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
    }

#if 0  /* Debugging */
  print_test_sads(sads_computed, image_size_macroblocks);
  write_sads_directly("sad-debug.bin",
		      ref_image->width / 16, ref_image->height / 16,
		      sads_computed);
#endif

  /* Free memory */
  free(sads_computed);
  free_image(ref_image);
  free_image(cur_image);

  pb_SwitchToTimer(&timers, pb_TimerID_NONE);
  pb_PrintTimerSet(&timers);
  pb_FreeParameters(params);

  return 0;
}
